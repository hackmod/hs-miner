#include "hip/hip_runtime.h"
#define DO_NOTHING(...) (__VA_ARGS__)
// Cuckoo Cycle, a memory-hard proof-of-work
// Copyright (c) 2013-2016 John Tromp

// The edge-trimming memory optimization is due to Dave Andersen
// http://da-data.blogspot.com/2014/03/a-public-review-of-cuckoo-cycle.html

#include <stdint.h>
#include <string.h>
#include "cuckoo.h"

// d(evice s)ipnode
#if (__CUDA_ARCH__  >= 320) // redefine ROTL to use funnel shifter, 3% speed gain

static __device__ __forceinline__ uint2 operator^ (uint2 a, uint2 b) { return make_uint2(a.x ^ b.x, a.y ^ b.y); }
static __device__ __forceinline__ void operator^= (uint2 &a, uint2 b) { a.x ^= b.x, a.y ^= b.y; }
static __device__ __forceinline__ void operator+= (uint2 &a, uint2 b) {
  asm("{\n\tadd.cc.u32 %0,%2,%4;\n\taddc.u32 %1,%3,%5;\n\t}\n\t"
    : "=r"(a.x), "=r"(a.y) : "r"(a.x), "r"(a.y), "r"(b.x), "r"(b.y));
}
#undef ROTL
__inline__ __device__ uint2 ROTL(const uint2 a, const int offset) {
  uint2 result;
  if (offset >= 32) {
    asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.x), "r"(a.y), "r"(offset));
    asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.y), "r"(a.x), "r"(offset));
  } else {
    asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.y), "r"(a.x), "r"(offset));
    asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.x), "r"(a.y), "r"(offset));
  }
  return result;
}
__device__ __forceinline__ uint2 vectorize(const uint64_t x) {
  uint2 result;
  asm("mov.b64 {%0,%1},%2; \n\t" : "=r"(result.x), "=r"(result.y) : "l"(x));
  return result;
}
__device__ __forceinline__ uint64_t devectorize(uint2 x) {
  uint64_t result;
  asm("mov.b64 %0,{%1,%2}; \n\t" : "=l"(result) : "r"(x.x), "r"(x.y));
  return result;
}
static __device__ node_t dipnode(siphash_keys &keys, edge_t nce, u32 uorv) {
  uint2 nonce = vectorize(2*nce + uorv);
  uint2 v0 = vectorize(keys.k0), v1 = vectorize(keys.k1), v2 = vectorize(keys.k2), v3 = vectorize(keys.k3) ^ nonce;
  SIPROUND; SIPROUND;
  v0 ^= nonce;
  v2 ^= vectorize(0xff);
  SIPROUND; SIPROUND; SIPROUND; SIPROUND;
  return devectorize(v0 ^ v1 ^ v2  ^ v3) & EDGEMASK;
}

#else

static __device__ node_t dipnode(siphash_keys &keys, edge_t nce, u32 uorv) {
  u64 nonce = 2*nce + uorv;
  u64 v0 = keys.k0, v1 = keys.k1, v2 = keys.k2, v3 = keys.k3 ^ nonce;
  SIPROUND; SIPROUND;
  v0 ^= nonce;
  v2 ^= 0xff;
  SIPROUND; SIPROUND; SIPROUND; SIPROUND;
  return (v0 ^ v1 ^ v2  ^ v3) & EDGEMASK;
}

#endif

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <set>

// algorithm parameters
#ifndef PART_BITS
// #bits used to partition edge set processing to save memory
// a value of 0 does no partitioning and is fastest
// a value of 1 partitions in two, making leancu_twice_set the
// same size as leancu_shrinkingset at about 33% slowdown
// higher values are not that interesting
#define PART_BITS 0
#endif

#ifndef IDXSHIFT
// we want sizeof(leancu_cuckoo_hash) == sizeof(leancu_twice_set), so
// CUCKOO_SIZE * sizeof(u64) == TWICE_WORDS * sizeof(u32)
// CUCKOO_SIZE * 2 == TWICE_WORDS
// (NNODES >> IDXSHIFT) * 2 == 2 * ONCE_BITS / 32
// NNODES >> IDXSHIFT == NEDGES >> PART_BITS >> 5
// IDXSHIFT == 1 + PART_BITS + 5
#define IDXSHIFT (PART_BITS + 6)
#endif

#define NODEBITS (EDGEBITS + 1)
#define NNODES (2 * NEDGES)
#define NODEMASK (NNODES-1)

// grow with cube root of size, hardly affected by trimming
#define MAXPATHLEN (8 << (NODEBITS/3))

#define freeCudaAllocs() \
  do { \
    if (ctx.alive.bits) hipFree(ctx.alive.bits); \
    if (ctx.nonleaf.bits) hipFree(ctx.nonleaf.bits); \
    if (device_ctx) hipFree(device_ctx);\
  } while (0)

#define checkCudaErrors(action) \
  do { \
    hipError_t c = (action); \
    if (c != hipSuccess) { \
      int32_t code = (int32_t)c; \
      if (code == 0) code = 1; \
      if (code < 0) code = -code; \
      freeCudaAllocs(); \
      return -code; \
    } \
  } while (0)
 
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  if (code != hipSuccess) {
    0;
    if (abort) exit(code);
  }
}

// set that starts out full and gets reset by threads on disjoint words
class leancu_shrinkingset {
public:
  u32 *bits;
  __device__ void reset(edge_t n) {
    bits[n/32] |= 1 << (n%32);
  }
  __device__ bool test(node_t n) const {
    return !((bits[n/32] >> (n%32)) & 1);
  }
  __device__ u32 block(node_t n) const {
    return ~bits[n/32];
  }
};

#define PART_MASK ((1 << PART_BITS) - 1)
#define ONCE_BITS (NEDGES >> PART_BITS)
#define TWICE_WORDS ((2 * ONCE_BITS) / 32)

class leancu_twice_set {
public:
  u32 *bits;
  __device__ void reset() {
    memset(bits, 0, TWICE_WORDS * sizeof(u32));
  }
  __device__ void set(node_t u) {
    node_t idx = u/16;
    u32 bit = 1 << (2 * (u%16));
    u32 old = atomicOr(&bits[idx], bit);
    u32 bit2 = bit<<1;
    if ((old & (bit2|bit)) == bit) atomicOr(&bits[idx], bit2);
  }
  __device__ u32 test(node_t u) const {
    return (bits[u/16] >> (2 * (u%16))) & 2;
  }
};

#define CUCKOO_SIZE (NNODES >> IDXSHIFT)
#define CUCKOO_MASK (CUCKOO_SIZE - 1)
// number of (least significant) key bits that survives leftshift by NODEBITS
#define KEYBITS (64-NODEBITS)
#define KEYMASK ((1L << KEYBITS) - 1)
#define MAXDRIFT (1L << (KEYBITS - IDXSHIFT))

class leancu_cuckoo_hash {
public:
  u64 *cuckoo;

  leancu_cuckoo_hash() {
    cuckoo = (u64 *)calloc(CUCKOO_SIZE, sizeof(u64));
    assert(cuckoo != 0);
  }
  ~leancu_cuckoo_hash() {
    free(cuckoo);
  }
  void set(node_t u, node_t v) {
    u64 niew = (u64)u << NODEBITS | v;
    for (node_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK) {
#ifdef ATOMIC
      u64 old = 0;
      if (cuckoo[ui].compare_exchange_strong(old, niew, std::memory_order_relaxed))
        return;
      if ((old >> NODEBITS) == (u & KEYMASK)) {
        cuckoo[ui].store(niew, std::memory_order_relaxed);
#else
      u64 old = cuckoo[ui];
      if (old == 0 || (old >> NODEBITS) == (u & KEYMASK)) {
        cuckoo[ui] = niew;
#endif
        return;
      }
    }
  }
  node_t operator[](node_t u) const {
    for (node_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK) {
#ifdef ATOMIC
      u64 cu = cuckoo[ui].load(std::memory_order_relaxed);
#else
      u64 cu = cuckoo[ui];
#endif
      if (!cu)
        return 0;
      if ((cu >> NODEBITS) == (u & KEYMASK)) {
        assert(((ui - (u >> IDXSHIFT)) & CUCKOO_MASK) < MAXDRIFT);
        return (node_t)(cu & NODEMASK);
      }
    }
  }
};

// arbitrary length of header hashed into siphash key


class leancu_cuckoo_ctx {
public:
  siphash_keys sip_keys;
  leancu_shrinkingset alive;
  leancu_twice_set nonleaf;
  int nthreads;

  leancu_cuckoo_ctx(const u32 n_threads) {
    nthreads = n_threads;
  }
  void setheadernonce(char* headernonce, uint32_t len, const u32 nonce) {
    ((u32 *)headernonce)[len/sizeof(u32)-1] = htole32(nonce); // place nonce at end
    setheader(headernonce, len, &sip_keys);
  }
};

__global__ void count_node_deg(leancu_cuckoo_ctx *ctx, u32 uorv, u32 part) {
  leancu_shrinkingset &alive = ctx->alive;
  leancu_twice_set &nonleaf = ctx->nonleaf;
  siphash_keys sip_keys = ctx->sip_keys; // local copy sip context; 2.5% speed gain
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  for (edge_t block = id*32; block < NEDGES; block += ctx->nthreads*32) {
    u32 alive32 = alive.block(block);
    for (edge_t nonce = block-1; alive32; ) { // -1 compensates for 1-based ffs
      u32 ffs = __ffs(alive32);
      nonce += ffs; alive32 >>= ffs;
      node_t u = dipnode(sip_keys, nonce, uorv);
      if ((u & PART_MASK) == part) {
        nonleaf.set(u >> PART_BITS);
      }
    }
  }
}

__global__ void kill_leaf_edges(leancu_cuckoo_ctx *ctx, u32 uorv, u32 part) {
  leancu_shrinkingset &alive = ctx->alive;
  leancu_twice_set &nonleaf = ctx->nonleaf;
  siphash_keys sip_keys = ctx->sip_keys;
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  for (edge_t block = id*32; block < NEDGES; block += ctx->nthreads*32) {
    u32 alive32 = alive.block(block);
    for (edge_t nonce = block-1; alive32; ) { // -1 compensates for 1-based ffs
      u32 ffs = __ffs(alive32);
      nonce += ffs; alive32 >>= ffs;
      node_t u = dipnode(sip_keys, nonce, uorv);
      if ((u & PART_MASK) == part) {
        if (!nonleaf.test(u >> PART_BITS)) {
          alive.reset(nonce);
        }
      }
    }
  }
}

static u32 path(leancu_cuckoo_hash &cuckoo, node_t u, node_t *us) {
  u32 nu;
  for (nu = 0; u; u = cuckoo[u]) {
    if (nu >= MAXPATHLEN) {
      while (nu-- && us[nu] != u) ;
      if (nu == ~0)
        0;
      else DO_NOTHING("illegal % 4d-cycle\n", MAXPATHLEN-nu);
      return (u32)-1;
    }
    us[nu++] = u;
  }
  return nu-1;
}

typedef std::pair<node_t,node_t> edge;

#include <unistd.h>
#include <stdbool.h>
#include "../common.h"

static int32_t lean_run(
  bool *running,
  uint32_t nthreads,
  uint32_t trims,
  uint32_t tpb,
  uint32_t nonce,
  uint32_t range,
  uint8_t *hdr,
  size_t hdrlen,
  uint8_t *target,
  uint8_t *solution,
  uint32_t *result,
  bool *match
) {
  int argc = 0;
  char *argv[1] = { NULL };
  uint8_t hash[32];
  uint8_t chash[32];
  uint32_t sol[PROOFSIZE];
  bool has_sol = false;
  uint32_t ononce = nonce;
  memset(hash, 0xff, 32);





  const char *header = "";
  int c;
  while ((c = getopt (argc, argv, "h:n:m:r:t:p:")) != -1) {
    switch (c) {
      case 'h':
        header = optarg;
        break;
      case 'n':
        nonce = atoi(optarg);
        break;
      case 'm':
        trims = atoi(optarg);
        break;
      case 't':
        nthreads = atoi(optarg);
        break;
      case 'p':
        tpb = atoi(optarg);
        break;
      case 'r':
        range = atoi(optarg);
        break;
    }
  }
  if (!tpb) // if not set, then default threads per block to roughly square root of threads
    for (tpb = 1; tpb*tpb < nthreads; tpb *= 2) ;

  0;
  if (range > 1)
    0;
  0;

  leancu_cuckoo_ctx ctx(nthreads);
  ctx.alive.bits = NULL;
  ctx.nonleaf.bits = NULL;
  ctx.nonleaf.bits = NULL;

  leancu_cuckoo_ctx *device_ctx = NULL;






  u64 edgeBytes = NEDGES/8, nodeBytes = TWICE_WORDS*sizeof(u32);
  checkCudaErrors(hipMalloc((void**)&ctx.alive.bits, edgeBytes));
  checkCudaErrors(hipMalloc((void**)&ctx.nonleaf.bits, nodeBytes));

  int edgeUnit=0, nodeUnit=0;
  u64 eb = edgeBytes, nb = nodeBytes;
  for (; eb >= 1024; eb>>=10) edgeUnit++;
  for (; nb >= 1024; nb>>=10) nodeUnit++;
  DO_NOTHING("Using %d%cB edge and %d%cB node memory.\n",
     (int)eb, " KMGT"[edgeUnit], (int)nb, " KMGT"[nodeUnit]);


  checkCudaErrors(hipMalloc((void**)&device_ctx, sizeof(leancu_cuckoo_ctx)));

  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  for (uint32_t r = 0; r < range; r++) {
    if (!*running)
      break;
 
    hipEventRecord(start, NULL);
    checkCudaErrors(hipMemset(ctx.alive.bits, 0, edgeBytes));
    ctx.setheadernonce((char *)hdr, hdrlen, nonce + r);
    hipMemcpy(device_ctx, &ctx, sizeof(leancu_cuckoo_ctx), hipMemcpyHostToDevice);
    for (u32 round=0; round < trims; round++) {
      for (u32 uorv = 0; uorv < 2; uorv++) {
        for (u32 part = 0; part <= PART_MASK; part++) {
          checkCudaErrors(hipMemset(ctx.nonleaf.bits, 0, nodeBytes));
          count_node_deg<<<nthreads/tpb,tpb >>>(device_ctx, uorv, part);
          kill_leaf_edges<<<nthreads/tpb,tpb >>>(device_ctx, uorv, part);
        }
      }
    }
  
    u64 *bits;
    bits = (u64 *)calloc(NEDGES/64, sizeof(u64));

    if (!bits) {
      freeCudaAllocs();
      return HS_ENOMEM;
    }

    if (!*running) {
      free(bits);
      freeCudaAllocs();
      return has_sol ? HS_SUCCESS : HS_ENOSOLUTION;
    }
 
    hipMemcpy(bits, ctx.alive.bits, (NEDGES/64) * sizeof(u64), hipMemcpyDeviceToHost);

    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    float duration;
    hipEventElapsedTime(&duration, start, stop);
    u32 cnt = 0;
    for (int i = 0; i < NEDGES/64; i++)
      cnt += __builtin_popcountll(~bits[i]);
    u32 load = (u32)(100L * cnt / CUCKOO_SIZE);
    DO_NOTHING("nonce %d: %d trims completed in %.3f seconds final load %d%%\n",
            nonce+r, trims, duration / 1000.0f, load);
  
    if (load >= 90) {
      0;
      free(bits);
      freeCudaAllocs();
      return HS_EMAXLOAD;
    }
  
    leancu_cuckoo_hash &cuckoo = *(new leancu_cuckoo_hash());
    node_t us[MAXPATHLEN], vs[MAXPATHLEN];
    for (edge_t block = 0; block < NEDGES; block += 64) {
      u64 alive64 = ~bits[block/64];
      for (edge_t nonce = block-1; alive64; ) { // -1 compensates for 1-based ffs
        u32 ffs = __builtin_ffsll(alive64);
        nonce += ffs; alive64 >>= ffs;
        node_t u0=sipnode(&ctx.sip_keys, nonce, 0), v0=sipnode(&ctx.sip_keys, nonce, 1);
        if (u0) {
          u32 nu = path(cuckoo, u0, us), nv = path(cuckoo, v0, vs);

          if ((int32_t)nu == -1 || (int32_t)nv == -1) {
            delete &cuckoo;
            free(bits);
            freeCudaAllocs();
            return HS_EBADPATH;
          }
 
          if (us[nu] == vs[nv]) {
            u32 min = nu < nv ? nu : nv;
            for (nu -= min, nv -= min; us[nu] != vs[nv]; nu++, nv++) ;
            u32 len = nu + nv + 1;
            0;
            if (len == PROOFSIZE) {
              0;
              std::set<edge> cycle;
              u32 n = 0;
              cycle.insert(edge(*us, *vs));
              while (nu--)
                cycle.insert(edge(us[(nu+1)&~1], us[nu|1])); // u's in even position; v's in odd
              while (nv--)
                cycle.insert(edge(vs[nv|1], vs[(nv+1)&~1])); // u's in odd position; v's in even
              for (edge_t blk = 0; blk < NEDGES; blk += 64) {
                u64 alv64 = ~bits[blk/64];
                for (edge_t nce = blk-1; alv64; ) { // -1 compensates for 1-based ffs
                  u32 ffs = __builtin_ffsll(alv64);
                  nce += ffs; alv64 >>= ffs;
                  edge e(sipnode(&ctx.sip_keys, nce, 0), sipnode(&ctx.sip_keys, nce, 1));
                  if (cycle.find(e) != cycle.end()) {
                    sol[n] = nce;
                    if (PROOFSIZE > 2)
                      cycle.erase(e);
                    n++;
                  }
                  if (ffs & 64) break; // can't shift by 64
                }
              }
              assert(n==PROOFSIZE);


              hs_hash_solution(sol, chash);

              if (memcmp(chash, hash, 32) <= 0) {
                *result = ononce + r;
                for (int32_t i = 0; i < PROOFSIZE; i++)
                  hs_write_u32(&solution[i * 4], sol[i]);
                memcpy(hash, chash, 32);
                has_sol = true;
              }

              if (memcmp(chash, target, 32) <= 0) {
                *match = true;
                delete &cuckoo;
                free(bits);
                freeCudaAllocs();
                return HS_SUCCESS;
              }

out:
              if (!*running) {
                delete &cuckoo;
                free(bits);
                freeCudaAllocs();
                return has_sol ? HS_SUCCESS : HS_ENOSOLUTION;
              }
 
              0;
            }
          } else if (nu < nv) {
            while (nu--)
              cuckoo.set(us[nu+1], us[nu]);
            cuckoo.set(u0, v0);
          } else {
            while (nv--)
              cuckoo.set(vs[nv+1], vs[nv]);
            cuckoo.set(v0, u0);
          }
        }
        if (ffs & 64) break; // can't shift by 64
      }
    }

    delete &cuckoo;
    free(bits);
  }

  freeCudaAllocs();

  return has_sol ? HS_SUCCESS : HS_ENOSOLUTION;
}
